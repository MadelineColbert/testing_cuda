#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <vector>
#include <cstdio>

void cusolver_problem() {
    hipsolverHandle_t handle;
    hipStream_t stream;

    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipsolverDnCreate(&handle);
    hipsolverSetStream(handle, stream);

    double* A;
    double* workspace;
    int* devipiv;
    int* devinfo;


    int m=0;
    int n=0;
    int lda=0;

    hipsolverDnDgetrf(handle, m,n, A, lda, workspace, devipiv, devinfo);

    hipStreamSynchronize(stream);

    hipsolverDnDestroy(handle);
    hipStreamDestroy(stream);

}

int main () {
    printf("Testing Make");
}