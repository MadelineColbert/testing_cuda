#include "hip/hip_runtime.h"
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <vector>
#include <cstdio>
#include <random>
#include "include/common.h"

using data_type=double;

struct csp_resp {
    int m;
    int n;
    std::vector<data_type> h_A;
    std::vector<data_type> workspace;
    std::vector<int> devipiv;
    int devinfo=0;

    csp_resp(int mv, int nv) 
        : m(mv), n(nv),
          h_A(mv*nv),
          devipiv(min(mv,nv)) {

    }
};

void cusolver_problem(csp_resp *test_in) {
    hipsolverHandle_t handle;
    hipStream_t stream;

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));
    CUSOLVER_CHECK(hipsolverSetStream(handle, stream));

    int lda=max(1,test_in->m);

    data_type *d_A = nullptr;
    data_type *d_workspace = nullptr;
    int *d_devipiv = nullptr;
    int *d_devinfo = nullptr;

    int malloc_size = test_in->m*test_in->n*sizeof(data_type);

    CUDA_CHECK(hipMalloc((void**)&d_A, malloc_size));
    CUDA_CHECK(hipMalloc((void**)&d_devipiv,  min(test_in->m,test_in->n)*sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_devinfo,  sizeof(int)));

    CUDA_CHECK(hipMemcpyAsync(d_A, test_in->h_A.data(), malloc_size, hipMemcpyHostToDevice, stream));

    int lwork=0;

    hipsolverDnDgetrf_bufferSize(handle, test_in->m, test_in->n, d_A, lda, &lwork);

    CUDA_CHECK(hipMalloc((void**)&d_workspace,  sizeof(data_type)*lwork));


    CUSOLVER_CHECK(hipsolverDnDgetrf(handle, test_in->m, test_in->n, d_A, lda, d_workspace, d_devipiv, d_devinfo));


    CUDA_CHECK(hipMemcpyAsync(test_in->h_A.data(), d_A, malloc_size, hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(test_in->workspace.data(), d_workspace, malloc_size, hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipMemcpyAsync(test_in->devipiv.data(), d_devipiv, min(test_in->m,test_in->n)*sizeof(int), hipMemcpyDeviceToHost, stream));
    
    CUDA_CHECK(hipMemcpyAsync(&(test_in->devinfo), d_devinfo, sizeof(int), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_workspace));
    CUDA_CHECK(hipFree(d_devinfo));
    CUDA_CHECK(hipFree(d_devipiv));

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));
    CUDA_CHECK(hipStreamDestroy(stream));

}

void generate_A(csp_resp *test_in){
    std::random_device rd;

    std::mt19937 gen(rd());

    std::uniform_int_distribution<> distrib(1,100);

    for (int i=0; i<(test_in->m * test_in->n); i++){
        int randomNumber = distrib(gen);
        test_in->h_A[i] = (data_type)randomNumber;   
    }
}

int main () {
    csp_resp test_in(10,10);
    generate_A(&test_in);
    cusolver_problem(&test_in);
    for (auto i : test_in.h_A) {
        printf("%f ", i);
    }
    printf("\n");
    for (auto i : test_in.workspace) {
        printf("%f ", i);
    }
    printf("\n");
    for (auto i : test_in.devipiv) {
        printf("%d ", i);
    }
}